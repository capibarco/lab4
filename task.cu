#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
__global__ void fill(double* matrixOld, double* matrixNew, int size)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	
	double fraction = 10.0 / (size - 1);
	
	matrixOld[i] = 10 + i * fraction;
	matrixOld[i * size] = 10 + i * fraction;
	matrixOld[size * i + size - 1] = 20 + i * fraction;
	matrixOld[size * (size - 1) + i] = 20 + i * fraction;

	matrixNew[i] = matrixOld[i];
	matrixNew[i * size] = matrixOld[i * size];
	matrixNew[size * i + size - 1] = matrixOld[size * i + size - 1];
	matrixNew[size * (size - 1) + i] = matrixOld[size * (size - 1) + i];
}
__global__ void calc(double* matrixOld, double* matrixNew, int n) 
{
    size_t i = blockIdx.x;
	size_t j = threadIdx.x;

	if((i > 0 && i < n-1) && (j > 0 && j < n-1))
	matrixNew[i * size + j] = 0.25 * (
					matrixOld[i * size + j - 1] +
					matrixOld[(i - 1) * size + j] +
					matrixOld[(i + 1) * size + j] +
					matrixOld[i * size + j + 1]);
}
__global__ void findError((double* matrixOld, double* matrixNew, double* matrixTmp, size_t n)
{
	if((i > 0 && i < n-1) && (j > 0 && j < n-1))
	{
		size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
		matrixTmp[idx] = matrixNew[idx] - matrixOld[idx];
	}
}
int main(int argc, char** argv)
{
	int cornerUL = 10;
	int cornerUR = 20;
	int cornerBR = 30;
	int cornerBL = 20;

	char* eptr;
	const double maxError = strtod((argv[1]), &eptr);
	const int size = atoi(argv[2]);
	const int maxIteration = atoi(argv[3]);
	const int toPrint = argc > 4 ? 1 : 0;

	int totalSize = size * size;

	double* matrixOld = (double*)calloc(totalSize, sizeof(double));
	double* matrixNew = (double*)calloc(totalSize, sizeof(double));
	double* matrixTmp = (double*)calloc(totalSize, sizeof(double));
	
    double* matrixOldD;
    double* matrixNewD;
    double* matrixTmpD;
	hipMalloc((void **)&matrixOldD, sizeof(double)*totalSize);
    hipMalloc((void **)&matrixNewD, sizeof(double)*totalSize);
    hipMalloc((void **)&matrixTmpD, sizeof(double)*totalSize);
	
	hipMemcpy(matrixOldD, matrixOld, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(matrixNewD, matrixNew, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(matrixTmpD, matrixTmp, sizeof(double), hipMemcpyHostToDevice);


	 int blockS, minGridSize = 128;
	 int maxSize = size; 
	 hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockS, calc, 0, totalSize);
	 dim3 blockSize(blockS, 1);
	 dim3 gridSize((size-1)/blockSize.x + 1, (size-1)/blockSize.y + 1);
	
	
 
    double* max_error, store=0;
    hipMalloc(&max_error, sizeof(double));

    size_t tempsize  = 0;
    hipcub::DeviceReduce::Max(store, tempsize, matrixTmpD, max_error, totalSize);
	hipMalloc(&store, tempsize);
	


	const double fraction = 10.0 / (size - 1);
	double errorNow = 1.0;
	int iterNow = 0;
	
	int result = 0;
	const double minus = -1;
	
	clock_t begin = clock();
	 fill<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, size);

while (errorNow > maxError && iterNow < maxIteration)
{
calc<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, size);
		
		if (iterNow % 100 == 0){	
			findError<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, matrixTmpD, size);
	 		
			 hipcub::DeviceReduce::Max(store, tempsize, matrixTmpD, max_error, totalSize);
        hipMemcpy(&errorNow, max_error, sizeof(double), hipMemcpyDeviceToHost);
		}

		double* t = matrixOldD;
		matrixOldD = matrixNewD;
		matrixNewD = c;

		iterNow++;
}
	clock_t end = clock();
	free(matrixOld);
	free(matrixNew);
	free(matrixTmp);
	hipFree(matrixOldD);
	hipFree(matrixNewD);
	hipFree(matrixTmpD);
	printf("iterations = %d, error = %lf, time = %lf\n", iterNow, errorNow, (double)(end - begin) / CLOCKS_PER_SEC);

	return 0;
}
