#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

__global__ void calc(double* matrixOld, double* matrixNew, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;//Индекс для обращения к элементу массива 
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;//Высчитывается из логики - Номер блока на размер блока(кол-во поток) плюс номер потока

    if((i > 0 && i < n-1) && (j > 0 && j < n-1))
        matrixNew[i * n + j] = 0.25 * (
                matrixOld[i * n + j - 1] +
                matrixOld[(i - 1) * n + j] +
                matrixOld[(i + 1) * n + j] +
                matrixOld[i * n + j + 1]);
}
__global__ void findError(double* matrixOld, double* matrixNew, double* matrixTmp, size_t n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;//Индекс для обращения к элементу массива 
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;//Высчитывается из логики - Номер блока на размер блока(кол-во поток) плюс номер потока
    if((i > 0 && i < n-1) && (j > 0 && j < n-1))
    {
        size_t idx = i * blockDim.x + j;
        matrixTmp[idx] = matrixNew[idx] - matrixOld[idx];
    }
}
__global__ void fill(double* arr, double* new_arr, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    arr[0] = new_arr[0] = 10;
    arr[n - 1]= new_arr[n - 1] = 20;
    arr[n * n - 1] = new_arr[n * n - 1] = 30;
    arr[n * (n - 1)] = new_arr[n * (n - 1)] = 20;
    if(i > 0 && i < n-1)
    {
        arr[i] = new_arr[i] = arr[0] + (10.0 / (n-1)) * i;
        arr[n*(n-1) + i] = new_arr[n*(n-1) + i] = arr[n - 1] + 10.0 / (n-1) * i;
        arr[n*i]= new_arr[n*i] = arr[0] + 10.0 / (n-1) * i;
        arr[n*i + n - 1] = new_arr[n*i + n - 1] = arr[n-1] + 10.0 / (n-1) * i;
    }
}
int main(int argc, char** argv)
{
    int cornerUL = 10;
    int cornerUR = 20;
    int cornerBR = 30;
    int cornerBL = 20;

    char* eptr;
    const double maxError = strtod((argv[1]), &eptr);
    const int size = atoi(argv[2]);
    const int maxIteration = atoi(argv[3]);
    const int toPrint = argc > 4 ? 1 : 0;

    int totalSize = size * size;

    double* matrixOld = (double*)calloc(totalSize, sizeof(double));
    double* matrixNew = (double*)calloc(totalSize, sizeof(double));
    double* matrixTmp = (double*)calloc(totalSize, sizeof(double));
/*
    const double fraction = 10.0 / (size - 1);
    for (int i = 0; i < size; i++)
    {
        matrixOld[i] = cornerUL + i * fraction;
        matrixOld[i * size] = cornerUL + i * fraction;
        matrixOld[size * i + size - 1] = cornerUR + i * fraction;
        matrixOld[size * (size - 1) + i] = cornerUR + i * fraction;

        matrixNew[i] = matrixOld[i];
        matrixNew[i * size] = matrixOld[i * size];
        matrixNew[size * i + size - 1] = matrixOld[size * i + size - 1];
        matrixNew[size * (size - 1) + i] = matrixOld[size * (size - 1) + i];
    }
*/

    double* matrixOldD = 0;
    double* matrixNewD = 0;
    double* matrixTmpD = 0;
    double* errorD =0;
    double* store = 0;

    hipMalloc((void **)&matrixOldD, sizeof(double)*totalSize);
    hipMalloc((void **)&matrixNewD, sizeof(double)*totalSize);
    hipMalloc((void **)&matrixTmpD, sizeof(double)*totalSize);
    hipMalloc((void **)&errorD, sizeof(double));
    size_t tempsize  = 0;
    hipcub::DeviceReduce::Max(store, tempsize, matrixTmpD, errorD, totalSize);
    hipMalloc((void**)&store, tempsize);


    hipMemcpy(matrixOldD, matrixOld, sizeof(double)*totalSize, hipMemcpyHostToDevice);
    hipMemcpy(matrixNewD, matrixNew, sizeof(double)*totalSize, hipMemcpyHostToDevice);


    int blockS, minGridSize = 128;
    int maxSize = size;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockS, calc, 0, totalSize);
    dim3 blockSize = dim3(32, 32);
    dim3 gridSize  = dim3(ceil(n/32.), ceil(n/32.));;

    double errorNow = 1.0;
    int iterNow = 0;

    
    clock_t begin = clock();
 fill<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, size);
    while (errorNow > maxError && iterNow < maxIteration)
    {
        calc<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, size);

        if (iterNow % 100 == 0)
        {
            findError<<<gridSize, blockSize>>>(matrixOldD, matrixNewD, matrixTmpD, size);

            hipcub::DeviceReduce::Max(store, tempsize, matrixTmpD, errorD, totalSize);
            hipMemcpy(&errorNow, errorD, sizeof(double), hipMemcpyDeviceToHost);
        }
        double* t = matrixOldD;
        matrixOldD = matrixNewD;
        matrixNewD = t;


        iterNow++;


    }
    clock_t end = clock();
    free(matrixOld);
    free(matrixNew);
    free(matrixTmp);
    hipFree(matrixOldD);
    hipFree(matrixNewD);
    hipFree(matrixTmpD);
    printf("iterations = %d, error = %lf, time = %lf\n", iterNow, errorNow, (double)(end - begin) / CLOCKS_PER_SEC);

    return 0;
}
